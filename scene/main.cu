
#include "hip/hip_runtime.h"
#include ""

#include "../src/simulation/Engine.h"
#include <iostream>

int main()
{


    MPM::Engine g_engine;
    g_engine.create();







    std::cout << "reach end of main\n";
    exit(EXIT_SUCCESS);
}

