#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "../src/simulation/Engine.h"
#include "../src/render/Renderer.hpp"

#include <iostream>

int main() {

  Renderer *renderer = Renderer::Builder()
      .init("MPM Engine") //TODO: window parameter
      .camera(glm::vec3(3., 3., 3), glm::vec3(0, 0, 0))
      .shader("../../src/render/shader/VertexShader.glsl", "../../src/render/shader/FragmentShader.glsl")
      .light(glm::vec3(5., 5., 5.),
             glm::vec3(1., 1., -1.),
             glm::vec3(1., 1., 1.),
             glm::vec3(0.1, 0.1, 0.1),
             glm::vec3(0, 0, 0))
      .build();

  auto handler = new InputHandler(renderer);

  GUIwrapper guiwrapper;
  guiwrapper
      .init(renderer->getWindow())
      .startGroup("Application Profile")
      .addWidgetText("Application average %.3f ms/frame (%.1f FPS)",
                     1000.0f / guiwrapper.getIO().Framerate,
                     guiwrapper.getIO().Framerate)
      .endGroup()
      .startGroup("Render Setting")
      .addWidgetColorEdit3("BackGround Color", renderer->m_background_color)
      .addWidgetColorEdit3("Default Particle Color", renderer->m_default_particle_color)
      .addWidgetSliderFloat("Particle Size", &renderer->m_particle_scale, 0.01f, 1.f)
      .addWidgetText("Camera Sensitivity")
      .addWidgetSliderFloat("Camera Translational Sensitivity", &renderer->getCamera().m_t_sensitivity, 0.01f, 0.2f)
      .addWidgetSliderFloat("Camera Rotational Sensitivity", &renderer->getCamera().m_r_sensitivity, 0.01f, 0.5f)
      .endGroup()
      .startGroup("Physics setting")
      .endGroup()
      .build();


  mpm::EngineConfig engine_config{

      true,
      mpm::FLIP,
      mpm::Explicit,
      mpm::Dense,
      mpm::Vec3i(64, 64, 64),
      1./64,
      1000,
  };
  mpm::Engine g_engine(engine_config);
//  g_engine.create(engine_config);
  g_engine.setGravity(mpm::Vec3f(0, 0, -9.8));

  mpm::Entity entity;
  unsigned int res = g_engine.getEngineConfig().m_gridResolution[0];
  float grid_dx = g_engine.getEngineConfig().m_gridCellSize;
  entity.loadCube(mpm::Vec3f(0.5, 0.5, 0.5), 0.5, 2 * (pow(res,3)/4));
  mpm::Particles particles(entity, mpm::WeaklyCompressibleWater, pow(grid_dx*0.5,3),1,"for debug"); //TODO: rho, initvol

  g_engine.addParticles(particles);
  int end_frame = 20000;
  int current_frame = 0;

  int deviceCount = 0;

  hipError_t e = hipGetDeviceCount(&deviceCount);
  e == hipSuccess ? deviceCount : -1;

  while (current_frame < end_frame && !glfwWindowShouldClose(renderer->getWindow())) { // hide glfw


    g_engine.integrate(7e-4);
    renderer->renderWithGUI(g_engine, guiwrapper);
    //renderer->getCamera().logCameraProperty();
    handler->handleInput();
    ++current_frame;

  }

  fmt::print("reach end of main\n");

  exit(EXIT_SUCCESS);
}

