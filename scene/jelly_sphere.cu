#include "hip/hip_runtime.h"



#include <Renderer.hpp>
#include <Engine.h>


#include <iostream>

Renderer* renderer = nullptr;
InputHandler* handler = nullptr;
GUIwrapper* gui = nullptr;
mpm::Engine* engine = nullptr;

float dt= 5e-4;
float x_data[10] = {0,1,2,3,4,5,6,7,8,9};
float y_data[10] = {25,29,21,17,15,13,11,9,7,5};
void initRenderer(){
  renderer = Renderer::Builder()
      .init("MPM Engine",1400,1400) //TODO: window parameter
      .camera(glm::vec3(3., 3., 3), glm::vec3(0, 0, 0))
      .shader("../../src/render/shader/VertexShader.glsl", "../../src/render/shader/FragmentShader.glsl")
      .light(glm::vec3(0.5, 0.5, 15),
             glm::vec3(1., 1., 1.),
             glm::vec3(0.1, 0.1, 0.1),
             glm::vec3(0, 0, 0))
      .build();
}
void initHandler(){
  handler = new InputHandler(renderer);
}
void initEngine(mpm::EngineConfig config){
  engine = new mpm::Engine(config);
  engine->setGravity(mpm::Vec3f(0, 0, 0));
  mpm::Entity entity;
  unsigned int res = engine->getEngineConfig().m_gridResolution[0];
  float grid_dx = engine->getEngineConfig().m_gridCellSize;
  entity.loadFromFile("../../assets/Sphere.bgeo");

  mpm::Particles particles(entity, mpm::MaterialType::CorotatedJelly, pow(grid_dx*0.5,3),1,mpm::Vec3f (0,0,0)); //TODO: rho, initvol

  float y_center=0.0;

  for(int i=0;i<particles.getParticleNum();i++){
    y_center+=particles.mParticleList[i].m_pos[1];
  }
  y_center/=particles.getParticleNum();
  for(int i=0;i<particles.getParticleNum();i++){
    if(particles.mParticleList[i].m_pos.y()>y_center) particles.mParticleList[i].m_vel[1] = 10.f;
    else particles.mParticleList[i].m_vel[1] = - 10.f;
  }
  engine->addParticles(particles);
  //fmt::print("CFL: {}\n", 1.0f*dt/engine->getEngineConfig().m_gridCellSize);

}
void initGui(){
  gui = new GUIwrapper();

  (*gui)
      .init(renderer->getWindow())
      .startGroup("Application Profile")
      .addWidgetText("Application average %.3f ms/frame (%.1f FPS)",
                    gui->m_average_time, gui->m_frame_rate)

      .endGroup()
      .startGroup("Render Setting")
      .addWidgetText("Color setting")
      .addWidgetColorEdit3("BackGround Color", renderer->m_background_color)
      .addWidgetColorEdit3("Particle Color", renderer->m_default_particle_color)
      .addWidgetSliderFloat("Particle Size", &renderer->m_particle_scale, 0.01f, 1.f)
      .addWidgetText("Camera Sensitivity")
      .addWidgetSliderFloat("Camera Translational Sensitivity", &renderer->getCamera().m_t_sensitivity, 0.01f, 0.2f)
      .addWidgetSliderFloat("Camera Rotational Sensitivity", &renderer->getCamera().m_r_sensitivity, 0.01f, 0.5f)
      .addWidgetInputFloat3("Camera Position", renderer->getCamera().getCameraPosFloatPtr())
      .addWidgetInputFloat3("Light Src Position", renderer->getLight().getLightScrPosFloatPtr())
      .endGroup()
      .startGroup("Physics setting")
      .addWidgetInputFloat3("Gravity setting",engine->getGravityFloatPtr())
      .addWidgetButton("Resume/Stop", [&](){
        if(engine->isRunning()){
          engine->stop();
        }else{
          engine->resume();
        }
      })
      .addWidgetText("%d Frame", engine->getCurrentFrame())
      .endGroup()
//      .startGroup("Energy plotting")
//      .startPlot("Integration profile")
//      .addPlotLine("Kinetic Energy", engine->getTimePtr(),engine->getParticleKineticEnergyPtr(),1000)
//      .endPlot()
//      .endGroup()
      .build();
}
void initDevice(){

  int deviceCount = 0;

  hipError_t e = hipGetDeviceCount(&deviceCount);
  e == hipSuccess ? deviceCount : -1;
}
void run(){
  while ( !glfwWindowShouldClose(renderer->getWindow())) { // hide glfw
        //engine->integrateWithCuda(8e-4);
    engine->integrate(dt);
    renderer->renderWithGUI((*engine), (*gui));



    handler->handleInput();

  }
}
int main() {


  initRenderer();
  initHandler();
  initEngine( mpm::EngineConfig {
      false,
      mpm::MLS,
      mpm::Explicit,
      mpm::Dense,
      mpm::Vec3i(64, 64, 64),
      2./64,
      1000,
      mpm::Device::GPU
  });
  initGui();

  run();


  fmt::print("reach end of main\n");

  exit(EXIT_SUCCESS);
}

