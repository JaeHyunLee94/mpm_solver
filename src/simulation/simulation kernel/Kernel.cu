#include "hip/hip_runtime.h"
//
// Created by test on 2022-03-07.
//

#include "../Particles.h"
#include "nvfunctional"
#include "hip/hip_runtime.h"
#include ""

namespace mpm {
#define SQR(x) ((x)*(x))

__global__ void integrateCuda(Particle &particle_ptr, Vec3f *grid_vel_ptr, Scalar *grid_mass_ptr, Scalar dt) {
  unsigned int taskId = threadIdx.x + blockIdx.x * blockDim.x;

}

__global__ void initGridCuda(Vec3f *d_grid_vel_ptr,
                             Scalar *d_grid_mass_ptr,
                             Scalar dt,
                             Scalar grid_dx,
                             unsigned int grid_dim_x,
                             unsigned int grid_dim_y,
                             unsigned int grid_dim_z) {
  unsigned int taskId = threadIdx.x + blockIdx.x * blockDim.x;

}

__global__ void p2gCuda(Particle *d_particles_ptr,
                        Vec3f *d_grid_vel_ptr,
                        Scalar *d_grid_mass_ptr,
                        Scalar dt,
                        Scalar grid_dx,
                        unsigned int grid_dim_x,
                        unsigned int grid_dim_y,
                        unsigned int grid_dim_z) {
  unsigned int taskId = threadIdx.x + blockIdx.x * blockDim.x;
  const Scalar grid_inv_dx = 1 / grid_dx;
  const Scalar _4_dt_invdx2 = 4.0f * dt * grid_inv_dx * grid_inv_dx;
  Particle &particle = d_particles_ptr[taskId];
  Vec3f Xp = particle.m_pos * grid_inv_dx;
  Vec3i base = (Xp - Vec3f(0.5f, 0.5f, 0.5f)).cast<int>();
  Vec3f fx = Xp - base.cast<Scalar>();
  //TODO: cubic function
  ////TODO: optimization candidate: so many constructor call?
  Vec3f w[3] = {0.5f * Vec3f(SQR(1.5f - fx[0]), SQR(1.5f - fx[1]), SQR(1.5f - fx[2])),
                Vec3f(0.75f - SQR(fx[0] - 1.0f),
                      0.75f - SQR(fx[1] - 1.0f),
                      0.75f - SQR(fx[2] - 1.0f)),
                0.5f * Vec3f(SQR(fx[0] - 0.5f), SQR(fx[1] - 0.5f), SQR(fx[2] - 0.5f))};


  ////TODO: optimization candidate: multiplication of matrix can be expensive.
  Mat3f cauchy_stress = particle.getStress(particle);//TODO: Std::bind


  Mat3f stress = cauchy_stress
      * (particle.m_Jp * particle.m_V0 * _4_dt_invdx2); ////TODO: optimization candidate: use inv_dx rather than dx
  Mat3f affine = stress + particle.m_mass * particle.m_Cp;
  //Scatter the quantity
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; ++j) {
      for (int k = 0; k < 3; ++k) {
        Vec3i offset{i, j, k};
        Scalar weight = w[i][0] * w[j][1] * w[k][2];
        Vec3f dpos = (offset.cast<Scalar>() - fx) * grid_dx;
        //i * _y_res * _z_res + j * _z_res + k
        Vec3i grid_index = base + offset;
        ////TODO: optimization candidate: assign dimension out side of the loop
        unsigned int idx =
            (grid_index[0] * grid_dim_y + grid_index[1]) * grid_dim_z
                + grid_index[2];
        Scalar mass_frag = weight * particle.m_mass;
        Vec3f momentum_frag = weight * (particle.m_mass * particle.m_vel + affine * dpos);

        atomicAdd(&d_grid_mass_ptr[idx], mass_frag);
        atomicAdd(&d_grid_vel_ptr[idx][0], momentum_frag[0]);
        atomicAdd(&d_grid_vel_ptr[idx][1], momentum_frag[1]);
        atomicAdd(&d_grid_vel_ptr[idx][2], momentum_frag[2]);
      }
    }
  }

}
__global__ void updateGridCuda(Vec3f *d_grid_vel_ptr,
                               Scalar *d_grid_mass_ptr,
                               Scalar dt,
                               Vec3f gravity,
                               Scalar grid_dx,
                               unsigned int grid_dim_x,
                               unsigned int grid_dim_y,
                               unsigned int grid_dim_z,
                               unsigned int bound) {
  unsigned int taskId = threadIdx.x + blockIdx.x * blockDim.x;
  if (d_grid_mass_ptr[taskId] > 0) {
    d_grid_mass_ptr[taskId] /= d_grid_mass_ptr[taskId];
    d_grid_vel_ptr[taskId] += dt * gravity;

    unsigned int xi = taskId / (grid_dim_y * grid_dim_z);
    unsigned int yi = (taskId - xi * grid_dim_y * grid_dim_z) / grid_dim_z;
    unsigned int zi = taskId - xi * grid_dim_y * grid_dim_z - yi * grid_dim_z;
    if (xi < bound && d_grid_vel_ptr[taskId][0] < 0) {
      d_grid_vel_ptr[taskId][0] = 0;
    } else if (xi > grid_dim_x - bound && d_grid_vel_ptr[taskId][0] > 0) {
      d_grid_vel_ptr[taskId][0] = 0;
    }
    if (yi < bound && d_grid_vel_ptr[taskId][1] < 0) {
      d_grid_vel_ptr[taskId][1] = 0;
    } else if (yi > grid_dim_y - bound && d_grid_vel_ptr[taskId][1] > 0) {
      d_grid_vel_ptr[taskId][1] = 0;
    }
    if (zi < bound && d_grid_vel_ptr[taskId][2] < 0) {
      d_grid_vel_ptr[taskId][2] = 0;
    } else if (zi > grid_dim_z - bound && d_grid_vel_ptr[taskId][2] > 0) {
      d_grid_vel_ptr[taskId][2] = 0;
    }

  }

}

__global__ void g2pCuda(Particle *d_particles_ptr, Vec3f *d_grid_vel_ptr, Scalar *d_grid_mass_ptr, Scalar dt) {
  unsigned int taskId = threadIdx.x + blockIdx.x * blockDim.x;

}

template<typename... Arguments>
void KernelLaunch(std::string &&tag, int gs, int bs, void(*f)(Arguments...), Arguments... args) {

}
}
